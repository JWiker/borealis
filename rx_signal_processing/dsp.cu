#include "hip/hip_runtime.h"
/*

Copyright 2017 SuperDARN Canada

See LICENSE for details

  \file dsp.cu
  This file contains the implementation for the all the needed GPU DSP work.
*/

#include "dsp.hpp"

#include <iostream>
#include <cstdlib>
#include <fstream>
#include <string>
#include <sstream>
#include <hip/hip_complex.h>
#include <chrono>
#include <thread>
#include <numeric>
#include <complex>
#include <eigen3/Eigen/Dense>
#include "utils/zmq_borealis_helpers/zmq_borealis_helpers.hpp"
#include "utils/signal_processing_options/signalprocessingoptions.hpp"
#include "utils/protobuf/sigprocpacket.pb.h"
#include "utils/protobuf/processeddata.pb.h"
#include "utils/shared_macros/shared_macros.hpp"
#include "filtering.hpp"
//TODO(keith): decide on handing gpu errors
//TODO(keith): potentially add multigpu support

//This keep postprocess local to this file.
namespace {
  /**
   * @brief      Sends an acknowledgment to the radar control and starts the timing after the
   *             RF samples have been copied.
   *
   * @param[in]  stream           CUDA stream this callback is associated with.
   * @param[in]  status           Error status of CUDA work in the stream.
   * @param[in]  processing_data  A pointer to the DSPCore associated with this CUDA stream.
   */
  void CUDART_CB initial_memcpy_callback_handler(hipStream_t stream, hipError_t status,
                          void *processing_data)
  {
    gpuErrchk(status);

    auto imc = [processing_data]()
    {
      auto dp = static_cast<DSPCore*>(processing_data);
      dp->send_ack();
      dp->start_decimate_timing();
      DEBUG_MSG(COLOR_RED("Finished initial memcpy handler for sequence #"
                 << dp->get_sequence_num() << ". Thread should exit here"));
    };

    std::thread start_imc(imc);
    start_imc.join();
  }


  /**
   * @brief      Drops samples contaminated by edge effects and filter roll off.
   *
   * @param      input_samples    The input samples.
   * @param      output_samples   The output samples.
   * @param      samps_per_stage  The number of output samples per stage.
   * @param      taps_per_stage   The number of filter taps per stage.
   * @param      dm_rates         The decimation rates of each stage.
   * @param[in]  num_antennas     The number of antennas.
   * @param[in]  num_freqs        The number of freqs.
   */
  void drop_bad_samples(hipComplex *input_samples, std::vector<hipComplex> &output_samples,
                        std::vector<uint32_t> &samps_per_stage,
                        std::vector<uint32_t> &taps_per_stage,
                        std::vector<uint32_t> &dm_rates,
                        uint32_t num_antennas, uint32_t num_freqs)
  {

    auto original_undropped_sample_count = samps_per_stage.back();
    auto original_samples_per_frequency = num_antennas * original_undropped_sample_count;

    // This accounts for the length of the filter extending past the length of input samples while
    // decimating.
    std::vector<uint32_t> bad_samples_per_stage;
    for (uint32_t i=0; i<dm_rates.size(); i++) {
      bad_samples_per_stage.push_back(uint32_t(std::floor(float(taps_per_stage[i]) /
                                                 float(dm_rates[i]))));
    }

    // Propagate the number of bad samples from the first stage through to the last stage.
    for (uint32_t i=1; i<bad_samples_per_stage.size(); i++) {
      bad_samples_per_stage[i] += std::ceil(float(bad_samples_per_stage[i-1])/(dm_rates[i]));
    }

    samps_per_stage.back() -= bad_samples_per_stage.back();
    auto samples_per_frequency = samps_per_stage.back() * num_antennas;

    output_samples.resize(num_freqs * samples_per_frequency);

    for (uint32_t freq_index=0; freq_index < num_freqs; freq_index++) {
      for (int i=0; i<num_antennas; i++){
        auto dest = output_samples.data() + (freq_index * samples_per_frequency) +
                    (i * samps_per_stage.back());
        auto src = input_samples + freq_index * (original_samples_per_frequency) +
                    (i * original_undropped_sample_count);
        auto num_bytes =  sizeof(hipComplex) * samps_per_stage.back();
        memcpy(dest, src, num_bytes);
      }
    }
  }

  /**
   * @brief      Beamforms the final samples
   *
   * @param      filtered_samples         A flat vector containing all the filtered samples for all
   *                                      RX frequencies.
   * @param      beamformed_samples_main  A vector where the beamformed and combined main array
   *                                      samples are placed.
   * @param      beamformed_samples_intf  A vector where the beamformed and combined intf array
   *                                      samples are placed.
   * @param      phases                   A flat vector of the phase delay offsets used to generate
   *                                      azimuthal directions. Phase offsets are complex
   *                                      exponential.
   * @param      num_main_ants            The number of main antennas.
   * @param      num_intf_ants            The number of intf antennas.
   * @param      beam_direction_counts    A vector containing the number of beam directions for each
   *                                      RX frequency.
   * @param      num_samples              The number of samples per antenna.
   *
   * This method extracts the offsets to the phases and samples needed for the beam directions of
   * each RX frequency. The Eigen library is then used to multiply the matrices to yield the final
   * beamformed samples. The main array and interferometer array are beamformed separately.
   */
  void beamform_samples(std::vector<hipComplex> &filtered_samples,
                        std::vector<hipComplex> &beamformed_samples_main,
                        std::vector<hipComplex> &beamformed_samples_intf,
                        std::vector<hipComplex> &phases, uint32_t num_main_ants,
                        uint32_t num_intf_ants, std::vector<uint32_t> beam_direction_counts,
                        uint32_t num_samples)
  {

    // Gonna make a lambda here to avoid repeated code. This is the main procedure that will
    // beamform the samples from offsets into the vectors.
    auto beamform_from_offsets = [&](hipComplex* samples_ptr,
                                      hipComplex* phases_ptr,
                                      hipComplex* result_ptr,
                                      uint32_t num_antennas, uint32_t num_beams)
    {

      // We work with hipComplex type for most DSP, but Eigen only knows the equivalent std lib type
      // so we cast to it for this context.
      auto samples_cast = reinterpret_cast<std::complex<float>*>(samples_ptr);
      auto phases_cast = reinterpret_cast<std::complex<float>*>(phases_ptr);

      // All we do here is map an existing set of memory to a structure that Eigen uses.
      Eigen::MatrixXcf samps = Eigen::Map<Eigen::Matrix<std::complex<float>,
                                                        Eigen::Dynamic,
                                                        Eigen::Dynamic,
                                                        Eigen::RowMajor>>(samples_cast,
                                                                          num_antennas,
                                                                          num_samples);
      Eigen::MatrixXcf phases = Eigen::Map<Eigen::Matrix<std::complex<float>,
                                                          Eigen::Dynamic,
                                                          Eigen::Dynamic,
                                                          Eigen::RowMajor>>(phases_cast,
                                                                            num_beams,
                                                                            num_antennas);

      // Result matrix has dimensions beams x num_samples. This means one set of samples for
      // each beam dir. Eigen overloads the * operator so we dont need to implement any matrix
      // work ourselves.
      auto result = phases * samps;

      // This piece of code just transforms the Eigen result back into our flat vector.
      auto beamformed_cast = reinterpret_cast<std::complex<float>*>(result_ptr);
      Eigen::Map<Eigen::Matrix<std::complex<float>, Eigen::Dynamic,
                                Eigen::Dynamic, Eigen::RowMajor>>(beamformed_cast, result.rows(),
                                                                  result.cols()) = result;
    };

    auto main_phase_offset = 0;
    auto main_results_offset = 0;

    // Now we calculate the offsets into the samples, phases, and results vector for each
    // RX frequency. Each RX frequency could have a different number of beams, so we increment
    // the phase and results offsets based off the accumulated number of beams. Once we have the
    // offsets, we can call the beamforming lambda.
    for (uint32_t rx_freq_num=0; rx_freq_num<beam_direction_counts.size(); rx_freq_num++) {

      auto num_beams = beam_direction_counts[rx_freq_num];

      // Increment to start of new frequency dataset.
      auto main_sample_offset = num_samples * (num_main_ants + num_intf_ants) * rx_freq_num;
      auto main_sample_ptr = filtered_samples.data() + main_sample_offset;

      auto main_phase_ptr = phases.data() + main_phase_offset;

      auto main_results_ptr = beamformed_samples_main.data() + main_results_offset;

      beamform_from_offsets(main_sample_ptr, main_phase_ptr, main_results_ptr,
                            num_main_ants, num_beams);

      // Only need to worry about beamforming the interferometer if its being used.
      if (num_intf_ants > 0) {

        // Skip the main array samples.
        auto intf_sample_offset = main_sample_offset + (num_samples * num_main_ants);
        auto intf_sample_ptr = filtered_samples.data() + intf_sample_offset;

        auto intf_phase_offset = main_phase_offset + (num_beams * num_main_ants);
        auto intf_phase_ptr = phases.data() + intf_phase_offset;

        // Result offsets will be the same. Each main and intf will have one set of samples for
        // each beam.
        auto intf_results_offset = main_results_offset;
        auto intf_results_ptr = beamformed_samples_intf.data() + intf_results_offset;

        beamform_from_offsets(intf_sample_ptr, intf_phase_ptr, intf_results_ptr,
                              num_intf_ants, num_beams);
      }

      //Possibly non uniform striding means we incremement the offset as we go.
      main_phase_offset += num_beams * (num_main_ants + num_intf_ants);
      main_results_offset += num_beams * num_samples;
    }

  }
  /**
   * @brief      Creates a data packet of processed data.
   *
   * @param      pd    A processeddata protobuf object.
   * @param      dp    A pointer to the DSPCore object with data to be extracted.
   *
   * This function extracts the processed data into a protobuf that data write can use.
   */
  void create_processed_data_packet(processeddata::ProcessedData &pd, DSPCore* dp)
  {

    std::vector<hipComplex> output_samples;

    auto samples_per_antenna = dp->get_samples_per_antenna();

    // create a new vector with the number of input rf samples included. Basically the equivalent
    // of a list concat in Python.
    std::vector<uint32_t> samps_per_stage;
    samps_per_stage.push_back(dp->get_num_rf_samples());
    samps_per_stage.insert(samps_per_stage.end(),
                           samples_per_antenna.begin(),
                           samples_per_antenna.end());

    auto filter_taps = dp->get_filter_taps();
    std::vector<uint32_t> taps_per_stage(filter_taps.size());
    for (uint32_t i=0; i<filter_taps.size(); i++) {
      taps_per_stage[i] = filter_taps[i].size();
    }

    auto filter_outputs_h = dp->get_filter_outputs_h();
    auto dm_rates = dp->get_dm_rates();
    drop_bad_samples(filter_outputs_h.back(), output_samples, samps_per_stage, taps_per_stage,
                     dm_rates, dp->get_num_antennas(), dp->get_rx_freqs().size());

    // For each antenna, for each frequency.
    auto num_samples_after_dropping = output_samples.size()/
                                      (dp->get_num_antennas()*dp->get_rx_freqs().size());




    auto total_beam_dirs = 0;
    auto beam_direction_counts = dp->get_beam_direction_counts();
    for(auto &beam_count : beam_direction_counts) {
      total_beam_dirs += beam_count;
    }

    std::vector<hipComplex> beamformed_samples_main(total_beam_dirs * num_samples_after_dropping);
    std::vector<hipComplex> beamformed_samples_intf(total_beam_dirs * num_samples_after_dropping);

    TIMEIT_IF_TRUE_OR_DEBUG(false,"Beamforming time: ",
      {
      auto beam_phases = dp->get_beam_phases();
      beamform_samples(output_samples, beamformed_samples_main, beamformed_samples_intf,
                        beam_phases,
                        dp->sig_options.get_main_antenna_count(),
                        dp->sig_options.get_interferometer_antenna_count(),
                        beam_direction_counts,
                        num_samples_after_dropping);
      }
    );

    // We have a lambda to extract the starting pointers of each set of output samples so that
    // we can use a consistent function to write either rf samples or stage data.
    auto make_ptrs_vec = [](hipComplex* output_p, uint32_t num_freqs, uint32_t num_antennas,
                              uint32_t num_samps_per_antenna)
    {
      auto stage_samps_per_set = num_antennas * num_samps_per_antenna;

      std::vector<std::vector<hipComplex*>> ptrs;
      for (uint32_t freq=0; freq<num_freqs; freq++) {
        std::vector<hipComplex*> stage_ptrs;
        for(uint32_t antenna=0; antenna<num_antennas; antenna++) {
          auto idx = freq * stage_samps_per_set + antenna * num_samps_per_antenna;
          stage_ptrs.push_back(output_p + idx);
        }
        ptrs.push_back(stage_ptrs);
      }

      return ptrs;
    };

    std::vector<std::vector<std::vector<hipComplex*>>> all_stage_ptrs;
    #ifdef ENGINEERING_DEBUG
      for (uint32_t i=0; i<filter_outputs_h.size(); i++) {
        auto ptrs = make_ptrs_vec(filter_outputs_h[i], dp->get_rx_freqs().size(),
                            dp->get_num_antennas(), samples_per_antenna[i]);
        all_stage_ptrs.push_back(ptrs);
      }
    #endif

    auto output_ptrs = make_ptrs_vec(output_samples.data(), dp->get_rx_freqs().size(),
                          dp->get_num_antennas(), num_samples_after_dropping);

    auto beamformed_offset = 0;
    for(uint32_t i=0; i<dp->get_rx_freqs().size(); i++) {
      auto dataset = pd.add_outputdataset();
      // This lambda adds the stage data to the processed data for debug purposes.
      auto add_debug_data = [dataset,i](std::string stage_name, std::vector<hipComplex*> &data_ptrs,
                                          uint32_t num_antennas, uint32_t num_samps_per_antenna)
      {
        auto debug_samples = dataset->add_debugsamples();

        debug_samples->set_stagename(stage_name);
        for (uint32_t j=0; j<num_antennas; j++){
          auto antenna_data = debug_samples->add_antennadata();
          for(uint32_t k=0; k<num_samps_per_antenna; k++) {
            auto antenna_samp = antenna_data->add_antennasamples();
            antenna_samp->set_real(data_ptrs[j][k].x);
            antenna_samp->set_imag(data_ptrs[j][k].y);
          } // close loop over samples
        } // close loop over antennas
      };

      // Add our beamformed IQ data to the processed data packet that gets sent to data_write.
      for (uint32_t beam_count=0; beam_count<beam_direction_counts[i]; beam_count++) {
        auto beam = dataset->add_beamformedsamples();
        beam->set_beamnum(beam_count);

        for (uint32_t sample=0; sample<num_samples_after_dropping; sample++){
          auto main_sample = beam->add_mainsamples();
          main_sample->set_real(beamformed_samples_main[beamformed_offset + sample].x);
          main_sample->set_imag(beamformed_samples_main[beamformed_offset + sample].y);

          if (dp->sig_options.get_interferometer_antenna_count() > 0) {
            auto intf_sample = beam->add_intfsamples();
            intf_sample->set_real(beamformed_samples_intf[beamformed_offset + sample].x);
            intf_sample->set_imag(beamformed_samples_intf[beamformed_offset + sample].y);
          }
        } // close loop over samples.
      } // close loop over beams.

      // Keep track of offsets as we move along frequencies. Different frequencies can have
      // different beams.
      beamformed_offset += beam_direction_counts[i];

      #ifdef ENGINEERING_DEBUG
        for (uint32_t j=0; j<all_stage_ptrs.size(); j++){
          auto stage_str = "stage_" + std::to_string(j);
          add_debug_data(stage_str, all_stage_ptrs[j][i], dp->get_num_antennas(),
            samples_per_antenna[j]);
        }
      #endif

      add_debug_data("output_ptrs", output_ptrs[i], dp->get_num_antennas(),
        num_samples_after_dropping);

      dataset->set_slice_id(dp->get_slice_ids()[i]);
      DEBUG_MSG("Created dataset for sequence #" << COLOR_RED(dp->get_sequence_num()));
    } // close loop over frequencies.

    pd.set_rf_samples_location(dp->get_shared_memory_name());
    pd.set_filter_rolloff_samples(dp->get_filter_rolloff_samples());
    pd.set_sequence_num(dp->get_sequence_num());
    pd.set_rx_sample_rate(dp->get_rx_rate());
    pd.set_output_sample_rate(dp->get_output_sample_rate());
    pd.set_processing_time(dp->get_decimate_timing());
    pd.set_initialization_time(dp->get_driver_initialization_time());
    pd.set_sequence_start_time(dp->get_sequence_start_time());
  }

  /**
   * @brief      Spawns the postprocessing work after all work in the CUDA stream is completed.
   *
   * @param[in]  stream           CUDA stream this callback is associated with.
   * @param[in]  status           Error status of CUDA work in the stream.
   * @param[in]  processing_data  A pointer to the DSPCore associated with this CUDA stream.
   *
   * The callback itself cannot call anything CUDA related as it may deadlock. It can, however
   * spawn a new thread and then exit gracefully, allowing the thread to do the work.
   */
  void CUDART_CB postprocess(hipStream_t stream, hipError_t status, void *processing_data)
  {

    gpuErrchk(status);

    auto pp = [processing_data]()
    {
      auto dp = static_cast<DSPCore*>(processing_data);

      dp->stop_timing();
      dp->send_timing();


      processeddata::ProcessedData pd;

      TIMEIT_IF_TRUE_OR_DEBUG(false, "Fill + send processed data time ",
        [&]() {
          create_processed_data_packet(pd,dp);
          dp->send_processed_data(pd);
        }()
      );

      DEBUG_MSG("Cuda kernel timing: " << COLOR_GREEN(dp->get_decimate_timing()) << "ms");
      DEBUG_MSG("Complete process timing: " << COLOR_GREEN(dp->get_total_timing()) << "ms");
      auto sq_num = dp->get_sequence_num();
      delete dp;

      DEBUG_MSG(COLOR_RED("Deleted DP in postprocess for sequence #" << sq_num
                  << ". Thread should terminate here."));
    };

    std::thread start_pp(pp);
    start_pp.detach();
  }

}


/**
 * @brief      Gets the properties of each GPU in the system.
 *
 * @return     The gpu properties.
 */
std::vector<hipDeviceProp_t> get_gpu_properties()
{
  std::vector<hipDeviceProp_t> gpu_properties;
  int num_devices = 0;

  gpuErrchk(hipGetDeviceCount(&num_devices));

  for(int i=0; i< num_devices; i++) {
      hipDeviceProp_t properties;
      gpuErrchk(hipGetDeviceProperties(&properties, i));
      gpu_properties.push_back(properties);
  }

  return gpu_properties;
}

/**
 * @brief      Prints the properties of each hipDeviceProp_t in the vector.
 *
 * @param[in]  gpu_properties  A vector of hipDeviceProp_t structs.
 *
 * More info on properties and calculations here:
 * https://devblogs.nvidia.com/parallelforall/how-query-device-properties-and-handle-errors-cuda-cc/
 */
void print_gpu_properties(std::vector<hipDeviceProp_t> gpu_properties) {
  for(auto i : gpu_properties) {
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "Device name: " << i.name);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max grid size x: " << i.maxGridSize[0]);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max grid size y: " << i.maxGridSize[1]);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max grid size z: " << i.maxGridSize[2]);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max threads per block: "
                << i.maxThreadsPerBlock);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max size of block dimension x: "
                << i.maxThreadsDim[0]);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max size of block dimension y: "
                << i.maxThreadsDim[1]);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max size of block dimension z: "
                << i.maxThreadsDim[2]);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Memory Clock Rate (GHz): "
                << i.memoryClockRate/1e6);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Memory Bus Width (bits): "
                << i.memoryBusWidth);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Peak Memory Bandwidth (GB/s): "
                << 2.0*i.memoryClockRate*(i.memoryBusWidth/8)/1.0e6);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max shared memory per block: "
                << i.sharedMemPerBlock);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Warpsize: " << i.warpSize);
  }
}


/**
 * @brief      Initializes the parameters needed in order to do asynchronous DSP processing.
 *
 * @param      ack_socket                  A pointer to the socket used for acknowledging when the
 *                                         transfer of RF samples has completed.
 * @param[in]  timing_socket               A pointer to the socket used for reporting GPU kernel
 *                                         timing.
 * @param      data_socket                 A pointer to the data socket used to sending processed
 *                                         data.
 * @param      sig_options                 The signal processing options.
 * @param[in]  sequence_num                The pulse sequence number for which will be acknowledged.
 * @param[in]  rx_rate                     The USRP sampling rate.
 * @param[in]  output_sample_rate          The final decimated output sample rate.
 * @param[in]  rx_freqs                    The receive freqs in Hz.
 * @param[in]  filter_taps                 The filter taps for each stage.
 * @param[in]  beam_phases                 The beam phases.
 * @param[in]  beam_direction_counts       The beam direction counts.
 * @param[in]  driver_initialization_time  The driver initialization time.
 * @param[in]  sequence_start_time         The sequence start time.
 * @param[in]  slice_ids                   The slice identifiers.
 * @param[in]  dm_rates                    The decimation rates.
 *
 * The constructor creates a new CUDA stream and initializes the timing events. It then opens the
 * shared memory with the received RF samples for a pulse sequence.
 */
DSPCore::DSPCore(zmq::socket_t *ack_socket, zmq::socket_t *timing_socket, zmq::socket_t *data_socket,
                  SignalProcessingOptions &sig_options, uint32_t sequence_num,
                  double rx_rate, double output_sample_rate, std::vector<double> rx_freqs,
                  std::vector<std::vector<float>> filter_taps,
                  std::vector<hipComplex> beam_phases, std::vector<uint32_t> beam_direction_counts,
                  double driver_initialization_time, double sequence_start_time,
                  std::vector<uint32_t> slice_ids, std::vector<uint32_t> dm_rates) :
  sequence_num(sequence_num),
  rx_rate(rx_rate),
  output_sample_rate(output_sample_rate),
  ack_socket(ack_socket),
  timing_socket(timing_socket),
  data_socket(data_socket),
  rx_freqs(rx_freqs),
  sig_options(sig_options),
  filter_taps(filter_taps),
  beam_phases(beam_phases),
  beam_direction_counts(beam_direction_counts),
  driver_initialization_time(driver_initialization_time),
  sequence_start_time(sequence_start_time),
  slice_ids(slice_ids),
  dm_rates(dm_rates)
{

  //https://devblogs.nvidia.com/parallelforall/gpu-pro-tip-cuda-7-streams-simplify-concurrency/
  gpuErrchk(hipStreamCreate(&stream));
  gpuErrchk(hipEventCreate(&initial_start));
  gpuErrchk(hipEventCreate(&kernel_start));
  gpuErrchk(hipEventCreate(&stop));
  gpuErrchk(hipEventCreate(&mem_transfer_end));
  gpuErrchk(hipEventRecord(initial_start, stream));

  shm = SharedMemoryHandler(random_string(20));


}

/**
 * @brief      Frees all associated pointers, events, and streams. Removes and deletes shared
 *             memory.
 */
DSPCore::~DSPCore()
{
  gpuErrchk(hipEventDestroy(initial_start));
  gpuErrchk(hipEventDestroy(kernel_start));
  gpuErrchk(hipEventDestroy(stop));
  gpuErrchk(hipFree(freqs_d));
  gpuErrchk(hipFree(rf_samples_d));
  gpuErrchk(hipFree(bp_filters_d));
  for (auto &filter : lp_filters_d) {
    gpuErrchk(hipFree(filter));
  }

  for (auto &filter_output : filter_outputs_d) {
    gpuErrchk(hipFree(filter_output));
  }

  for (auto &filter_output : filter_outputs_h) {
    gpuErrchk(hipHostFree(filter_output));
  }

  gpuErrchk(hipStreamDestroy(stream));

}

/**
 * @brief      Allocates device memory for the RF samples and then copies them to device.
 *
 * @param[in]  total_antennas         The total number of antennas.
 * @param[in]  num_samples_needed     The number of samples needed from each antenna ringbuffer.
 * @param[in]  extra_samples          The number of extra samples needed for filter propagation.
 * @param[in]  offset_to_first_pulse  Offset from sequence start to center of first pulse.
 * @param[in]  time_zero              The time the driver began collecting samples. seconds since
 *                                    epoch.
 * @param[in]  start_time             The start time of the pulse sequence. seconds since epoch.
 * @param[in]  ringbuffer_size        The ringbuffer size in number of samples.
 * @param      ringbuffer_ptrs_start  A vector of pointers to the start of each antenna ringbuffer.
 *
 * Samples are being stored in a shared memory ringbuffer. This function calculates where to index
 * into the ringbuffer for samples and copies them to the gpu. This function will also copy the
 * samples to a shared memory section that data write, or another process can access in order to
 * work with the raw RF samples.
 */
void DSPCore::allocate_and_copy_rf_samples(uint32_t total_antennas, uint32_t num_samples_needed,
                                int64_t extra_samples, uint32_t offset_to_first_pulse,
                                double time_zero, double start_time,
                                uint64_t ringbuffer_size,
                                std::vector<hipComplex*> &ringbuffer_ptrs_start)
{


  size_t rf_samples_size = total_antennas * num_samples_needed * sizeof(hipComplex);
  shm.create_shr_mem(rf_samples_size);
  gpuErrchk(hipMalloc(&rf_samples_d, rf_samples_size));

  auto sample_time_diff = start_time - time_zero;
  auto sample_in_time = (sample_time_diff * rx_rate) +
                      offset_to_first_pulse -
                      extra_samples;
  auto start_sample = int64_t(std::fmod(sample_in_time, ringbuffer_size));

  if ((start_sample) < 0) {
   start_sample += ringbuffer_size;
  }

  if ((start_sample + num_samples_needed) > ringbuffer_size) {
    for (uint32_t i=0; i<total_antennas; i++) {
      auto first_piece = ringbuffer_size - start_sample;
      auto second_piece = num_samples_needed - first_piece;

      auto first_dest = rf_samples_d + (i*num_samples_needed);
      auto second_dest = rf_samples_d + (i*num_samples_needed) + (first_piece);

      auto first_src = ringbuffer_ptrs_start[i] + start_sample;
      auto second_src = ringbuffer_ptrs_start[i];

      gpuErrchk(hipMemcpyAsync(first_dest, first_src, first_piece * sizeof(hipComplex),
                                 hipMemcpyHostToDevice, stream));
      gpuErrchk(hipMemcpyAsync(second_dest, second_src, second_piece * sizeof(hipComplex),
                                 hipMemcpyHostToDevice, stream));

      auto mem_cast = static_cast<hipComplex*>(shm.get_shrmem_addr());
      auto first_dest_h = mem_cast + (i*num_samples_needed);
      auto second_dest_h = mem_cast + (i*num_samples_needed) + (first_piece);

      memcpy(first_dest_h, first_src, first_piece * sizeof(hipComplex));
      memcpy(second_dest_h, second_src, second_piece * sizeof(hipComplex));
    }

  }
  else {
    for (uint32_t i=0; i<total_antennas; i++) {
      auto dest = rf_samples_d + (i*num_samples_needed);
      auto src = ringbuffer_ptrs_start[i] + start_sample;

      gpuErrchk(hipMemcpyAsync(dest, src, num_samples_needed * sizeof(hipComplex),
        hipMemcpyHostToDevice, stream));

      auto mem_cast = static_cast<hipComplex*>(shm.get_shrmem_addr());
      auto dest_h = mem_cast + (i*num_samples_needed);
      memcpy(dest_h, src, num_samples_needed * sizeof(hipComplex));
    }
  }


}

/**
 * @brief      Allocates device memory for the filtering frequencies and then copies them to device.
 *
 * @param      freqs      A pointer to the filtering freqs.
 * @param[in]  num_freqs  The number of freqs.
 */
void DSPCore::allocate_and_copy_frequencies(void *freqs, uint32_t num_freqs) {
  size_t freqs_size = num_freqs * sizeof(double);
  gpuErrchk(hipMalloc(&freqs_d, freqs_size));
  gpuErrchk(hipMemcpyAsync(freqs_d, freqs, freqs_size, hipMemcpyHostToDevice, stream));
}

/**
 * @brief      Allocate and copy bandpass filters for all rx freqs to gpu.
 *
 * @param      taps        A pointer to the filter taps.
 * @param[in]  total_taps  The total amount of filter taps.
 */
void DSPCore::allocate_and_copy_bandpass_filters(void *taps, uint32_t total_taps)
{
  size_t bp_filters_size = total_taps * sizeof(hipComplex);
  gpuErrchk(hipMalloc(&bp_filters_d, bp_filters_size));
  gpuErrchk(hipMemcpyAsync(bp_filters_d, taps, bp_filters_size, hipMemcpyHostToDevice, stream));
}

/**
 * @brief      Allocate and copy a lowpass filter to the gpu.
 *
 * @param      taps        A pointer to the filter taps.
 * @param[in]  total_taps  The total amount of filter taps.
 */
void DSPCore::allocate_and_copy_lowpass_filter(void *taps, uint32_t total_taps)
{
  hipComplex *ptr_d;
  lp_filters_d.push_back(ptr_d);

  size_t filter_size = total_taps * sizeof(hipComplex);
  gpuErrchk(hipMalloc(&lp_filters_d.back(), filter_size));
  gpuErrchk(hipMemcpyAsync(lp_filters_d.back(), taps, filter_size, hipMemcpyHostToDevice, stream));

}

/**
 * @brief      Gets the last filter output d.
 *
 * @return     The last filter output d.
 */
hipComplex* DSPCore::get_last_filter_output_d()
{
  return filter_outputs_d.back();
}

/**
 * @brief      Gets the last pointer stored in the lowpass filters vector.
 *
 * @return     The last lowpass filter pointer inserted into the vector.
 */
hipComplex* DSPCore::get_last_lowpass_filter_d() {
  return lp_filters_d.back();
}

/**
 * @brief      Gets the samples per antenna vector. Vector contains an element for each stage.
 *
 * @return     The samples per antenna vector.
 */
std::vector<uint32_t> DSPCore::get_samples_per_antenna() {
  return samples_per_antenna;
}

/**
 * @brief      The vector containing vectors of filter taps for each stage.
 *
 * @return     The filter taps vectors for each stage.
 */
std::vector<std::vector<float>> DSPCore::get_filter_taps() {
  return filter_taps;
}

/**
 * @brief      Allocate a filter output on the GPU.
 *
 * @param[in]  num_output_samples  The number output samples
 */
void DSPCore::allocate_output(uint32_t num_output_samples)
{
  hipComplex *ptr_d;
  filter_outputs_d.push_back(ptr_d);
  size_t output_size = num_output_samples * sizeof(hipComplex);
  gpuErrchk(hipMalloc(&filter_outputs_d.back(), output_size));

}

/**
 * @brief      Allocate a host pointer for decimation stage output and then copy data.
 *
 * @param[in]  num_output_samples  The number output samples needed.
 * @param      output_d            The device pointer from which to copy from.
 */
void DSPCore::allocate_and_copy_host(uint32_t num_output_samples, hipComplex *output_d)
{
  hipComplex *ptr_h;
  filter_outputs_h.push_back(ptr_h);

  size_t output_size = num_output_samples * sizeof(hipComplex);
  gpuErrchk(hipHostMalloc(&filter_outputs_h.back(), output_size));
  gpuErrchk(hipMemcpyAsync(filter_outputs_h.back(), output_d,
        output_size, hipMemcpyDeviceToHost,stream));

}

/**
 * @brief      Stops the timers that the constructor starts.
 */
void DSPCore::stop_timing()
{
  gpuErrchk(hipEventRecord(stop, stream));
  gpuErrchk(hipEventSynchronize(stop));

  gpuErrchk(hipEventElapsedTime(&total_process_timing_ms, initial_start, stop));
  gpuErrchk(hipEventElapsedTime(&decimate_kernel_timing_ms, kernel_start, stop));
  gpuErrchk(hipEventElapsedTime(&mem_time_ms, initial_start, mem_transfer_end));
  RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "Cuda memcpy time: "
    << COLOR_GREEN(mem_time_ms) << "ms");
  RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "Decimate time: "
    << COLOR_GREEN(decimate_kernel_timing_ms) << "ms");

}

/**
 * @brief      Sends the GPU kernel timing to the radar control.
 *
 * The timing here is used as a rate limiter, so that the GPU doesn't become backlogged with data.
 * If the GPU is overburdened, this will result in less averages, but the system wont crash.
 */
void DSPCore::send_timing()
{
  sigprocpacket::SigProcPacket sp;
  sp.set_kerneltime(decimate_kernel_timing_ms);
  sp.set_sequence_num(sequence_num);

  std::string s_msg_str;
  sp.SerializeToString(&s_msg_str);

  auto request = RECV_REQUEST(*timing_socket, sig_options.get_brian_dspend_identity());
  SEND_REPLY(*timing_socket, sig_options.get_brian_dspend_identity(), s_msg_str);

  DEBUG_MSG(COLOR_RED("Sent timing after processing with sequence #" << sequence_num));

}


/**
 * @brief      Add the postprocessing callback to the stream.
 *
 * This function allocates the host space needed for filter stage data and then copies the data
 * from GPU into the allocated space. Certain DSPCore members needed for post processing are
 * assigned such as the rx freqs, the number of rf samples, the total antennas and the vector
 * of samples per antenna(each stage).
 */
void DSPCore::cuda_postprocessing_callback(std::vector<double> freqs, uint32_t total_antennas,
                                            uint32_t num_samples_rf,
                                            std::vector<uint32_t> samples_per_antenna,
                                            std::vector<uint32_t> total_output_samples,
                                            uint32_t extra_samples)
{
  #ifdef ENGINEERING_DEBUG
    for (uint32_t i=0; i<filter_outputs_d.size()-1; i++) {
      allocate_and_copy_host(total_output_samples[i], filter_outputs_d[i]);
    }
  #endif

  allocate_and_copy_host(total_output_samples.back(), filter_outputs_d.back());

  rx_freqs = freqs;
  num_rf_samples = num_samples_rf;
  num_antennas = total_antennas;
  this->samples_per_antenna = samples_per_antenna;
  filter_rolloff_samples = extra_samples;

  gpuErrchk(hipStreamAddCallback(stream, postprocess, this, 0));

  DEBUG_MSG(COLOR_RED("Added stream callback for sequence #" << sequence_num));
}

/**
 * @brief      Sends the acknowledgment to the radar control that the RF samples have been
 *             transfered.
 *
 * RF samples of one pulse sequence can be transfered asynchronously while samples of another are
 * being processed. This means that it is possible to start running a new pulse sequence in the
 * driver as soon as the samples are copied. The asynchronous nature means only timing constraint
 * is the time needed to run the GPU kernels for decimation.
 */
void DSPCore::send_ack()
{
  sigprocpacket::SigProcPacket sp;
  sp.set_sequence_num(sequence_num);

  std::string s_msg_str;
  sp.SerializeToString(&s_msg_str);

  auto request = RECV_REQUEST(*ack_socket, sig_options.get_brian_dspbegin_identity());
  SEND_REPLY(*ack_socket, sig_options.get_brian_dspbegin_identity(), s_msg_str);

  DEBUG_MSG(COLOR_RED("Sent ack after copy for sequence_num #" << sequence_num));
}

/**
 * @brief      Sends a processed data packet to data write.
 *
 * @param      pd    A processeddata protobuf object.
 */
void DSPCore::send_processed_data(processeddata::ProcessedData &pd)
{
  std::string p_msg_str;
  pd.SerializeToString(&p_msg_str);

  SEND_REPLY(*data_socket, sig_options.get_dw_dsp_identity(), p_msg_str);

  DEBUG_MSG(COLOR_RED("Send processed data to data_write for sequence #" << sequence_num));
}


/**
 * @brief      Starts the timing before the GPU kernels execute.
 *
 */
void DSPCore::start_decimate_timing()
{
  gpuErrchk(hipEventRecord(kernel_start, stream));
  gpuErrchk(hipEventRecord(mem_transfer_end,stream));
}

/**
 * @brief      Adds the callback to the CUDA stream to acknowledge the RF samples have been copied.
 *
 */
void DSPCore::initial_memcpy_callback()
{
  gpuErrchk(hipStreamAddCallback(stream, initial_memcpy_callback_handler, this, 0));
}


/**
 * @brief      Gets the device pointer to the RF samples.
 *
 * @return     The RF samples device pointer.
 */
hipComplex* DSPCore::get_rf_samples_p(){
  return rf_samples_d;
}

/**
 * @brief      Gets the host pointer to the RF samples.
 *
 * @return     The rf samples host pointer.
 */
std::vector<hipComplex> DSPCore::get_rf_samples_h() {
  return rf_samples_h;
}

/**
 * @brief      Gets the device pointer to the receive frequencies.
 *
 * @return     The frequencies device pointer.
 */
double* DSPCore::get_frequencies_p() {
  return freqs_d;
}


/**
 * @brief      Gets the bandpass filters device pointer.
 *
 * @return     The bandpass filter pointer.
 */
hipComplex* DSPCore::get_bp_filters_p(){
  return bp_filters_d;
}

/**
 * @brief      Gets the vector of decimation rates.
 *
 * @return     The dm rates.
 */
std::vector<uint32_t> DSPCore::get_dm_rates()
{
  return dm_rates;
}

/**
 * @brief      Gets the vector of host side filter outputs.
 *
 * @return     The filter outputs host vector.
 */
std::vector<hipComplex*> DSPCore::get_filter_outputs_h()
{
  return filter_outputs_h;
}

/**
 * @brief      Get the vector of host side frequencies.
 *
 * @return     The receive freqs vector.
 */
std::vector<double> DSPCore::get_rx_freqs()
{
  return rx_freqs;
}
/**
 * @brief      Gets the CUDA stream this DSPCore's work is associated to.
 *
 * @return     The CUDA stream.
 */
hipStream_t DSPCore::get_cuda_stream(){
  return stream;
}

/**
 * @brief      Gets the total GPU process timing in milliseconds.
 *
 * @return     The total process timing.
 */
float DSPCore::get_total_timing()
{
  return total_process_timing_ms;
}

/**
 * @brief      Gets the total decimation timing in milliseconds.
 *
 * @return     The decimation timing.
 */
float DSPCore::get_decimate_timing()
{
  return decimate_kernel_timing_ms;
}

/**
 * @brief      Gets the number of antennas.
 *
 * @return     The number of antennas.
 */
uint32_t DSPCore::get_num_antennas()
{
  return num_antennas;
}

/**
 * @brief      Gets the number of rf samples.
 *
 * @return     The number of rf samples.
 */
uint32_t DSPCore::get_num_rf_samples()
{
  return num_rf_samples;
}

/**
 * @brief      Gets the filter rolloff samples.
 *
 * @return     The filter rolloff samples.
 */
uint32_t DSPCore::get_filter_rolloff_samples()
{
  return filter_rolloff_samples;
}

/**
 * @brief      Gets the sequence number.
 *
 * @return     The sequence number.
 */
uint32_t DSPCore::get_sequence_num()
{
  return sequence_num;
}

/**
 * @brief      Gets the rx sample rate.
 *
 * @return     The rx sampling rate (samples per second).
 */
double DSPCore::get_rx_rate()
{
  return rx_rate;
}

/**
 * @brief      Gets the output sample rate.
 *
 * @return     The output decimated and filtered rate (samples per second).
 */
double DSPCore::get_output_sample_rate()
{
  return output_sample_rate;
}

/**
 * @brief     Gets the vector of beam phases.
 *
 * @return    The beam phases.
 */
std::vector<hipComplex> DSPCore::get_beam_phases()
{
  return beam_phases;
}

/**
 * @brief     Gets the vector of beam direction counts for each RX frequency.
 *
 * @return    The beam direction counts.
 */
std::vector<uint32_t> DSPCore::get_beam_direction_counts()
{
  return beam_direction_counts;
}

/**
 * @brief     Gets the name of the shared memory section.
 *
 * @return    The shared memory name string.
 */
std::string DSPCore::get_shared_memory_name()
{
  return shm.get_region_name();
}

/**
 * @brief      Gets the driver initialization timestamp.
 *
 * @return     The driver initialization timestamp.
 */
double DSPCore::get_driver_initialization_time()
{
  return driver_initialization_time;
}

/**
 * @brief      Gets the sequence start timestamp.
 *
 * @return     The sequence start timestamp.
 */
double DSPCore::get_sequence_start_time()
{
  return sequence_start_time;
}

/**
 * @brief      Gets the vector of slice identifiers.
 *
 * @return     The vector of slice identifiers.
 */
std::vector<uint32_t> DSPCore::get_slice_ids()
{
  return slice_ids;
}
